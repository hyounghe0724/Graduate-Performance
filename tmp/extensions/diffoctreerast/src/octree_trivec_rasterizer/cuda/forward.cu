#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
namespace cg = cooperative_groups;

#include "config.h"
#include "auxiliary.h"
#include "data_structure.h"
#include "api.h"


/**
 * Helper function to find the highest bit set in an integer.
 * 
 * @param n Integer.
 * @return Highest bit set.
*/
static uint32_t getHigherMsb(uint32_t n) {
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}


/**
 * Forward pass for converting the input spherical harmonics coefficients of each voxel to color.
 * 
 * @param deg Degree of the spherical harmonics coefficients.
 * @param max_coeffs Maximum number of coefficients.
 * @param mean 3D points.
 * @param campos Camera position.
 * @param sh spherical harmonics coefficients.
 * @param color Output color.
 */
static __device__ void computeColorFromSH(int deg, int max_coeffs, int trivec_rank, const glm::vec3* mean, glm::vec3 campos, const float* sh, float* color) {
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = *mean;
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	for (int tc = 0; tc < trivec_rank; tc++)
		for (int ch = 0; ch < CHANNELS; ch++)
			color[tc * CHANNELS + ch] = SH_C0 * sh[tc * max_coeffs * CHANNELS + ch];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		float coeff[3] = { -SH_C1 * y, SH_C1 * z, -SH_C1 * x };
		for (int tc = 0; tc < trivec_rank; tc++)
			for (int ch = 0; ch < CHANNELS; ch++)
				color[tc * CHANNELS + ch] +=
					coeff[0] * sh[tc * max_coeffs * CHANNELS + 1 * CHANNELS + ch] +
					coeff[1] * sh[tc * max_coeffs * CHANNELS + 2 * CHANNELS + ch] +
					coeff[2] * sh[tc * max_coeffs * CHANNELS + 3 * CHANNELS + ch];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			float coeff[5] = { SH_C2[0] * xy, SH_C2[1] * yz, SH_C2[2] * (2.0f * zz - xx - yy), SH_C2[3] * xz, SH_C2[4] * (xx - yy) };
			for (int tc = 0; tc < trivec_rank; tc++)
				for (int ch = 0; ch < CHANNELS; ch++)
					color[tc * CHANNELS + ch] +=
						coeff[0] * sh[tc * max_coeffs * CHANNELS + 4 * CHANNELS + ch] +
						coeff[1] * sh[tc * max_coeffs * CHANNELS + 5 * CHANNELS + ch] +
						coeff[2] * sh[tc * max_coeffs * CHANNELS + 6 * CHANNELS + ch] +
						coeff[3] * sh[tc * max_coeffs * CHANNELS + 7 * CHANNELS + ch] +
						coeff[4] * sh[tc * max_coeffs * CHANNELS + 8 * CHANNELS + ch];

			if (deg > 2)
			{
				float coeff[7] = {
					SH_C3[0] * y * (3.0f * xx - yy),
					SH_C3[1] * xy * z,
					SH_C3[2] * y * (4.0f * zz - xx - yy),
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy),
					SH_C3[4] * x * (4.0f * zz - xx - yy),
					SH_C3[5] * z * (xx - yy),
					SH_C3[6] * x * (xx - 3.0f * yy)
				};
				for (int tc = 0; tc < trivec_rank; tc++)
					for (int ch = 0; ch < CHANNELS; ch++)
						color[tc * CHANNELS + ch] +=
							coeff[0] * sh[tc * max_coeffs * CHANNELS + 9 * CHANNELS + ch] +
							coeff[1] * sh[tc * max_coeffs * CHANNELS + 10 * CHANNELS + ch] +
							coeff[2] * sh[tc * max_coeffs * CHANNELS + 11 * CHANNELS + ch] +
							coeff[3] * sh[tc * max_coeffs * CHANNELS + 12 * CHANNELS + ch] +
							coeff[4] * sh[tc * max_coeffs * CHANNELS + 13 * CHANNELS + ch] +
							coeff[5] * sh[tc * max_coeffs * CHANNELS + 14 * CHANNELS + ch] +
							coeff[6] * sh[tc * max_coeffs * CHANNELS + 15 * CHANNELS + ch];
			}
		}
	}
}


/**
 * Compute the morton code for a 3D point based on the camera position and the depth of the voxel.
 * 
 * @param pos Position of the point.
 * @param campos Camera position.
 * @param depth Depth of the voxel.
 */
static __device__ uint32_t computeMortonCode(float3 pos, float3 campos, uint8_t depth) {
	uint32_t mul = 1 << MAX_TREE_DEPTH;
	uint32_t xcode = (uint32_t)(pos.x * mul);
	uint32_t ycode = (uint32_t)(pos.y * mul);
	uint32_t zcode = (uint32_t)(pos.z * mul);
	uint32_t cxcode = (uint32_t)(campos.x * mul);
	uint32_t cycode = (uint32_t)(campos.y * mul);
	uint32_t czcode = (uint32_t)(campos.z * mul);
	uint32_t xflip = 0, yflip = 0, zflip = 0;
	bool done = false;
	for (int i = 1; i <= MAX_TREE_DEPTH && !done; i++)
	{
		xflip |= ((xcode >> (MAX_TREE_DEPTH - i + 1) << 1) < (cxcode >> (MAX_TREE_DEPTH - i))) ? (1 << (MAX_TREE_DEPTH - i)) : 0;
		yflip |= ((ycode >> (MAX_TREE_DEPTH - i + 1) << 1) < (cycode >> (MAX_TREE_DEPTH - i))) ? (1 << (MAX_TREE_DEPTH - i)) : 0;
		zflip |= ((zcode >> (MAX_TREE_DEPTH - i + 1) << 1) < (czcode >> (MAX_TREE_DEPTH - i))) ? (1 << (MAX_TREE_DEPTH - i)) : 0;
		done = i == depth;
	}
	xcode ^= xflip;
	ycode ^= yflip;
	zcode ^= zflip;
	return expandBits(xcode) | (expandBits(ycode) << 1) | (expandBits(zcode) << 2);
}


/**
 * Preprocess input 3D points
 */
static __global__ void preprocess(
	const int num_nodes,
	const int active_sh_degree,
	const int num_sh_coefs,
	const int trivec_rank,
	const float* positions,
	const float* shs,
	const uint8_t* tree_depths,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int width,
	const int height,
	const float* aabb,
	float* colors,
	int4* bboxes,
	const dim3 grid,
	uint32_t* tiles_touched,
	uint32_t* morton_codes
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= num_nodes)
		return;

	// Initialize bboxes and touched tiles to 0. If this isn't changed,
	// this voxel will not be processed further.
	bboxes[idx] = { 0, 0, 0, 0 };
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_orig = {
		positions[3 * idx] * aabb[3] + aabb[0],
		positions[3 * idx + 1] * aabb[4] + aabb[1],
		positions[3 * idx + 2] * aabb[5] + aabb[2]
	};
	float3 p_view;
	if (!in_frustum(idx, p_orig, viewmatrix, projmatrix, p_view))
		return;

	// Project 8 vertices of the voxel to screen space to find the
	// bounding box of the projected points.
	float nsize = powf(2.0f, -(float)tree_depths[idx]) * scale_modifier;
	float3 scale = { aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize };
	int4 bbox = get_bbox(p_orig, scale, projmatrix, width, height);
	uint2 rect_min, rect_max;
	getRect(bbox, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (shs) {
		computeColorFromSH(
			active_sh_degree, num_sh_coefs, trivec_rank,
			(glm::vec3*)&p_orig, *cam_pos,
			shs + idx * trivec_rank * num_sh_coefs * CHANNELS,
			colors + idx * trivec_rank * CHANNELS
		);
	}

	// Calculate view-dependent morton code for sorting.
	float3 pos = { positions[3 * idx], positions[3 * idx + 1], positions[3 * idx + 2] };
	float3 ncampos = {
		max(0.0f, min(1.0f, (cam_pos->x - aabb[0]) / aabb[3])),
		max(0.0f, min(1.0f, (cam_pos->y - aabb[1]) / aabb[4])),
		max(0.0f, min(1.0f, (cam_pos->z - aabb[2]) / aabb[5]))
	};
	uint32_t morton_code = computeMortonCode(pos, ncampos, tree_depths[idx]);

	// Store some useful helper data for the next steps.
	bboxes[idx] = bbox;
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
	morton_codes[idx] = morton_code;
}


/**
 * Generates one key/value pair for all voxel / tile overlaps. 
 * Run once per voxel (1:N mapping).
 * 
 * @param P Number of points.
 * @param points_xy 2D points.
 * @param depths Depths of points.
 * @param offsets Offsets for writing keys/values.
 * @param keys_unsorted Unsorted keys.
 * @param values_unsorted Unsorted values.
 * @param radii Radii of points.
 * @param grid Grid size.
 */
static __global__ void duplicateWithKeys(
	int P,
	const uint32_t* morton_codes,
	const uint32_t* offsets,
	uint64_t* keys_unsorted,
	uint32_t* values_unsorted,
	int4* bboxes,
	dim3 grid
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible voxels
	if (bboxes[idx].w > 0)
	{
		// Find this voxel's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;
		getRect(bboxes[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the voxel. Sorting the values 
		// with this key yields voxel IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= morton_codes[idx];
				keys_unsorted[off] = key;
				values_unsorted[off] = idx;
				off++;
			}
		}
	}
}


/**
 * Check keys to see if it is at the start/end of one tile's range in the full sorted list. If yes, write start/end of this tile.
 * 
 * @param L Number of points.
 * @param point_list_keys List of keys.
 * @param ranges Ranges of tiles.
 */
static __global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}


static __device__ void sample_trivec(
	const float* trivec,
	const int trivec_dim,
	const float* densities,
	const float* colors,
	const float density_shift,
	const int used_rank,
	const float3& p,
	const float3& voxel_min,
	const float3& voxel_max,
	float& out_density,
	float* out_color
) {
	// Compute linear interpolation weights
	float3 _p = {
		(p.x - voxel_min.x) / (voxel_max.x - voxel_min.x) * trivec_dim - 0.5f,
		(p.y - voxel_min.y) / (voxel_max.y - voxel_min.y) * trivec_dim - 0.5f,
		(p.z - voxel_min.z) / (voxel_max.z - voxel_min.z) * trivec_dim - 0.5f
	};
	int3 _ip = { 
		min(trivec_dim - 2, max(0, (int)_p.x)),
		min(trivec_dim - 2, max(0, (int)_p.y)),
		min(trivec_dim - 2, max(0, (int)_p.z))
	};
	float3 w = { _p.x - _ip.x, _p.y - _ip.y, _p.z - _ip.z };

	float _density;
	for (int i = 0; i < used_rank; i++) {
		_density = lerp(trivec[TRIVEC_X_CH(trivec_dim, i) + _ip.x], trivec[TRIVEC_X_CH(trivec_dim, i) + _ip.x + 1], w.x)
				 * lerp(trivec[TRIVEC_Y_CH(trivec_dim, i) + _ip.y], trivec[TRIVEC_Y_CH(trivec_dim, i) + _ip.y + 1], w.y)
				 * lerp(trivec[TRIVEC_Z_CH(trivec_dim, i) + _ip.z], trivec[TRIVEC_Z_CH(trivec_dim, i) + _ip.z + 1], w.z);
		out_density += densities[i] * _density;
		for (int j = 0; j < CHANNELS; j++)
			out_color[j] += colors[CHANNELS * i + j] * _density;
	}
	for (int j = 0; j < CHANNELS; j++)
		out_color[j] = sigmoid(out_color[j]);
	out_density = softplus(out_density - density_shift * 10) * min(1 / (1 - density_shift), 25.0f);
}


/**
 * Main rasterization method. Collaboratively works on one tile per
 * block, each thread treats one pixel. Alternates between fetching 
 * and rasterizing data.
 * 
 * @param ranges Ranges of voxel instances for each tile.
 * @param point_list List of voxel instances.
 * @param W Width of the image.
 * @param H Height of the image.
 * @param bg_color Background color.
 * @param cam_pos Camera position.
 * @param tan_fovx Tangent of the horizontal field of view.
 * @param tan_fovy Tangent of the vertical field of view.
 * @param viewmatrix View matrix.
 * @param aabb Axis-aligned bounding box.
 * @param positions Centers of octree nodes.
 * @param trivecs Trivec features of octree nodes.
 * @param trivec_rank Trivec rank.
 * @param trivec_dim Trivec dimension.
 * @param densities Densities of octree nodes.
 * @param density_shift Density shift.
 * @param colors Colors of octree nodes.
 * @param tree_depths Depths of octree nodes.
 * @param scale_modifier Scale modifier.
 * @param random_image Random image.
 * @param n_contrib Number of contributors.
 * @param out_color Output color.
 * @param out_depth Output depth.
 * @param out_alpha Output alpha.
 * @param out_percent_depth Output percent depth.
 */
static __global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
render(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const int W,
	const int H,
	const float* __restrict__ bg_color,
	const float3* cam_pos,
	const float tan_fovx,
	const float tan_fovy,
	const float* __restrict__ viewmatrix,
	const float* __restrict__ aabb,
	const float* __restrict__ positions,
	const float* __restrict__ trivecs,
	const int trivec_rank,
	const int trivec_dim,
	const float* __restrict__ densities,
	const float density_shift,
	const float* __restrict__ colors,
	const int used_rank,
	const uint8_t* __restrict__ tree_depths,
	const float scale_modifier,
	const float* __restrict__ random_image,
	const float* __restrict__ colors_overwrite,
	uint32_t* __restrict__ n_contrib,
	uint32_t* __restrict__ t_contrib,
	float* __restrict__ out_color,
	float* __restrict__ out_depth,
	float* __restrict__ out_alpha,
	float* __restrict__ out_percent_depth

	// DEBUG
	// ,int dbg_ray_id,
	// float* __restrict__ dbg_position,
	// float* __restrict__ dbg_density,
	// float* __restrict__ dbg_color,
	// float* __restrict__ dbg_weight
) {
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;

	// Get ray direction and origin for this pixel.
	const float2 jt_pix = { pix.x + random_image[pix_id * 3 + 0], pix.y + random_image[pix_id * 3 + 1] };
	float3 ray_dir = getRayDir(jt_pix, W, H, tan_fovx, tan_fovy, viewmatrix);
	float depth_div = 1.0f / sqrtf(ray_dir.x * ray_dir.x + ray_dir.y * ray_dir.y + ray_dir.z * ray_dir.z);
	ray_dir.x *= depth_div;
	ray_dir.y *= depth_div;
	ray_dir.z *= depth_div;

	// Check if this thread is associated with a valid pixel or outside.
	const bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + PREFETCH_BUFFER_SIZE - 1) / PREFETCH_BUFFER_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for shared memory
	const int trivec_size = TRIVEC_SIZE(trivec_dim, trivec_rank);
	extern __shared__ char shared_mem[];
	uint32_t* collected_ids = (uint32_t*)shared_mem;
	float3* collected_scales = (float3*)(collected_ids + PREFETCH_BUFFER_SIZE);
	float3* collected_xyz = (float3*)(collected_scales + PREFETCH_BUFFER_SIZE);
	float* collected_trivecs = (float*)(collected_xyz + PREFETCH_BUFFER_SIZE);
	float* collected_densities = (float*)(collected_trivecs + PREFETCH_BUFFER_SIZE * trivec_size);
	float* collected_colors = (float*)(collected_densities + PREFETCH_BUFFER_SIZE * trivec_rank);

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	uint32_t last_t = 0;
	float C[CHANNELS] = { 0 };
	float D = 0, PD = 0;
	const float jitter = random_image[pix_id * 3 + 2];

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= PREFETCH_BUFFER_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-voxel data from global to shared
		#ifndef ASYNC_GLOBAL_TO_SHARED
		for (int j = 0; j < PREFETCH_BUFFER_SIZE; j++)
		{
			int progress = i * PREFETCH_BUFFER_SIZE + j;
			if (range.x + progress < range.y)
			{
				int coll_id = point_list[range.x + progress];
				float nsize = powf(2.0f, -(float)tree_depths[coll_id]) * scale_modifier;
				if (block.thread_rank() == 0) {
					collected_ids[j] = coll_id;
					collected_scales[j] = { aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize };
					collected_xyz[j] = {
						positions[3 * coll_id] * aabb[3] + aabb[0],
						positions[3 * coll_id + 1] * aabb[4] + aabb[1],
						positions[3 * coll_id + 2] * aabb[5] + aabb[2]
					};
				}
				for (int ch = block.thread_rank(); ch < trivec_size; ch += BLOCK_SIZE)
					collected_trivecs[j * trivec_size + ch] = trivecs[coll_id * trivec_size + ch];
				for (int ch = block.thread_rank(); ch < trivec_rank; ch += BLOCK_SIZE)
					collected_densities[j * trivec_rank + ch] = densities[coll_id * trivec_rank + ch];
				for (int ch = block.thread_rank(); ch < trivec_rank * CHANNELS; ch += BLOCK_SIZE)
					collected_colors[j * trivec_rank * CHANNELS + ch] = colors[coll_id * trivec_rank * CHANNELS + ch];
			}
		}
		block.sync();
		#else
		for (int j = 0; j < PREFETCH_BUFFER_SIZE; j++)
		{
			int progress = i * PREFETCH_BUFFER_SIZE + j;
			if (range.x + progress < range.y)
			{
				int coll_id = point_list[range.x + progress];
				float nsize = powf(2.0f, -(float)tree_depths[coll_id]) * scale_modifier;
				if (block.thread_rank() == 0) {
					collected_ids[j] = coll_id;
					collected_scales[j] = { aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize };
					collected_xyz[j] = {
						positions[3 * coll_id] * aabb[3] + aabb[0],
						positions[3 * coll_id + 1] * aabb[4] + aabb[1],
						positions[3 * coll_id + 2] * aabb[5] + aabb[2]
					};
				}
				cg::memcpy_async(block, collected_trivecs + j * trivec_size, trivecs + coll_id * trivec_size, trivec_size * sizeof(float));
				cg::memcpy_async(block, collected_densities + j * trivec_rank, densities + coll_id * trivec_rank, trivec_rank * sizeof(float));
				cg::memcpy_async(block, collected_colors + j * trivec_rank * CHANNELS, colors + coll_id * trivec_rank * CHANNELS, trivec_rank * CHANNELS * sizeof(float));
			}
		}
		cg::wait(block);
		block.sync();
		#endif

		// Iterate over current batch
		for (int j = 0; !done && j < min(PREFETCH_BUFFER_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Get ray-voxel intersection
			float3 p = collected_xyz[j];
			float3 scale = collected_scales[j];
			float3 voxel_min = { p.x - 0.5f * scale.x, p.y - 0.5f * scale.y, p.z - 0.5f * scale.z };
			float3 voxel_max = { p.x + 0.5f * scale.x, p.y + 0.5f * scale.y, p.z + 0.5f * scale.z };
			float2 itsc = get_ray_voxel_intersection(*cam_pos, ray_dir, voxel_min, voxel_max);
			float itsc_dist = (itsc.y >= itsc.x) ? itsc.y - itsc.x : -1.0f;
			if (itsc_dist <= 0.0f)
				continue;

			// Volume rendering
			float step = (0.5f / trivec_dim) * scale.x;
			int t_start = (int)ceil(itsc.x / step - jitter);
			int t_end = (int)floor(itsc.y / step - jitter);
			for (int t = t_start; t <= t_end; t++) {	
				float z = (t + jitter) * step;

				// Sample trivec
				float density = 0;
				float color[CHANNELS] = { 0 };
				p = { cam_pos->x + z * ray_dir.x, cam_pos->y + z * ray_dir.y, cam_pos->z + z * ray_dir.z };
				sample_trivec(
					collected_trivecs + j * trivec_size, trivec_dim,
					collected_densities + j * trivec_rank,
					collected_colors + j * trivec_rank * CHANNELS,
					density_shift, used_rank,
					p, voxel_min, voxel_max,
					density, color
				);
				if (colors_overwrite != nullptr)
					for (int k = 0; k < CHANNELS; k++)
						color[k] = colors_overwrite[CHANNELS * collected_ids[j] + k];

				// Accumulate
				float alpha = min(1 - exp(-density * step), 0.999f);
				const float weight = alpha * T;
				for (int k = 0; k < CHANNELS; k++)
					C[k] += color[k] * weight;
				D += z * weight;

				T *= 1 - alpha;
				if (PD == 0 && T < 0.5f)
					PD = z;
				last_t = t;
				if (T < 0.001f)
					break;

				// DEBUG
				// if (pix_id == dbg_ray_id) {
				// 	dbg_position[3 * t + 0] = p.x;
				// 	dbg_position[3 * t + 1] = p.y;
				// 	dbg_position[3 * t + 2] = p.z;
				// 	dbg_density[t] = density;
				// 	for (int k = 0; k < CHANNELS; k++)
				// 		dbg_color[CHANNELS * t + k] = color[k];
				// 	dbg_weight[t] = weight;
				// }
			}

			// Keep track of last range entry to update this pixel.
			last_contributor = contributor;

			// If we have accumulated enough, we can stop
			if (T < 0.001f)
				done = true;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		n_contrib[pix_id] = last_contributor;
		t_contrib[pix_id] = last_t;
		for (int k = 0; k < CHANNELS; k++)
			out_color[k * H * W + pix_id] = C[k] + T * bg_color[k];
		out_depth[pix_id] = D * depth_div;
		out_alpha[pix_id] = 1.0f - T;
		out_percent_depth[pix_id] = PD * depth_div;
	}
}

int OctreeTrivecRasterizer::CUDA::forward(
	std::function<char*(size_t)> geometryBuffer,
	std::function<char*(size_t)> binningBuffer,
	std::function<char*(size_t)> imageBuffer,
	const int num_nodes,
    const int active_sh_degree,
    const int num_sh_coefs,
	const float* background,
    const int width,
    const int height,
    const float* aabb,
    const float* positions,
    const float* trivecs,
	const int trivec_rank,
	const int trivec_dim,
	const float* densities,
	const float density_shift,
    const float* shs,
    const float* colors,
    const int used_rank,
    const uint8_t* depths,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
    const float* cam_pos,
	const float tan_fovx,
    const float tan_fovy,
    const float* random_image,
	const float* colors_overwrite,
    float* out_color,
    float* out_depth,
    float* out_alpha,
	float* out_percent_depth

	// DEBUG
	// ,int dbg_ray_id,
	// float* dbg_position,
	// float* dbg_density,
	// float* dbg_color,
	// float* dbg_weight
) {
	DEBUG_PRINT("Starting forward pass\n");
	DEBUG_PRINT("    - Number of nodes: %d\n", num_nodes);
	DEBUG_PRINT("    - Image size: %d x %d\n", width, height);
	DEBUG_PRINT("    - Trivec rank: %d\n", trivec_rank);
	DEBUG_PRINT("    - Trivec dimension: %d\n", trivec_dim);

	// Parrallel config (2D grid of 2D blocks)
	dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Allocate buffers for auxiliary info for points and pixels
	DEBUG_PRINT("Allocating buffers\n");
	size_t buffer_size;
	char* buffer_ptr;
	buffer_size = required<GeometryState>(num_nodes, trivec_rank);
	DEBUG_PRINT("    - Geometry buffer size: %zu\n", buffer_size);
	buffer_ptr = geometryBuffer(buffer_size);
	GeometryState geomState = GeometryState::fromChunk(buffer_ptr, num_nodes, trivec_rank);
	buffer_size = required<ImageState>(width * height);
	DEBUG_PRINT("    - Image buffer size: %zu\n", buffer_size);
	buffer_ptr = imageBuffer(buffer_size);
	ImageState imgState = ImageState::fromChunk(buffer_ptr, width * height);

	// Run preprocessing kernel
	DEBUG_PRINT("Calling preprocess kernel\n");
	CHECK_CUDA(preprocess<<<(num_nodes+255)/256, 256>>>(
		num_nodes, active_sh_degree, num_sh_coefs, trivec_rank,
		positions, shs, depths, scale_modifier,
		viewmatrix, projmatrix, (glm::vec3*)cam_pos,
		width, height, aabb, geomState.colors,
		geomState.bboxes, grid, geomState.tiles_touched, geomState.morton_codes
	));

	// Compute prefix sum over full list of touched tile counts by voxels
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(
		geomState.scanning_space, geomState.scan_size,
		geomState.tiles_touched, geomState.point_offsets, num_nodes
	));

	// Retrieve total number of voxel instances to launch
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + num_nodes - 1, sizeof(int), hipMemcpyDeviceToHost));
	if (num_rendered == 0)
		return 0;

	// Allocate buffer for binning state
	DEBUG_PRINT("Allocating binning buffer\n");
	DEBUG_PRINT("    - Number of rendered nodes: %d\n", num_rendered);
	buffer_size = required<BinningState>(num_rendered);
	DEBUG_PRINT("    - Binning buffer size: %zu\n", buffer_size);
	buffer_ptr = binningBuffer(buffer_size);
	BinningState binningState = BinningState::fromChunk(buffer_ptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated voxel indices to be sorted
	DEBUG_PRINT("Calling duplicateWithKeys kernel\n");
	CHECK_CUDA(duplicateWithKeys<<<(num_nodes+255)/256, 256>>>(
		num_nodes, geomState.morton_codes, geomState.point_offsets,
		binningState.point_list_keys_unsorted, binningState.point_list_unsorted,
		geomState.bboxes, grid
	));

	// Sort complete list of (duplicated) voxel indices by keys
	int bit = getHigherMsb(grid.x * grid.y);
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space, binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit
	));

	// Identify start and end of per-tile workloads in sorted list
	CHECK_CUDA(hipMemset(imgState.ranges, 0, grid.x * grid.y * sizeof(uint2)));
	CHECK_CUDA(identifyTileRanges<<<(num_rendered+255)/256, 256>>>(
		num_rendered, binningState.point_list_keys, imgState.ranges
	));

	// Let each tile blend its range of voxels independently in parallel
	const float* color_ptr = (shs) ? geomState.colors : colors;
	size_t used_memory = PREFETCH_BUFFER_SIZE * (
		sizeof(uint32_t) + 															// collected_ids
		sizeof(float3) + 															// collected_scales
		sizeof(float3) + 															// collected_xyz
		TRIVEC_SIZE(trivec_dim, trivec_rank) * sizeof(float) + 						// collected_trivecs
		trivec_rank * sizeof(float) + 												// collected_densities
		trivec_rank * CHANNELS * sizeof(float) 										// collected_colors
	);
	DEBUG_PRINT("Calling render kernel\n");
	DEBUG_PRINT("    - Used shared memory: %zu\n", used_memory);
	CHECK_CUDA(render<<<grid, block, used_memory>>>(
		imgState.ranges, binningState.point_list,
		width, height, background,
		(float3*)cam_pos, tan_fovx, tan_fovy, viewmatrix, aabb,
		positions, trivecs, trivec_rank, trivec_dim, densities, density_shift, color_ptr, used_rank, depths, scale_modifier, random_image, colors_overwrite,
		imgState.n_contrib, imgState.t_contrib, out_color, out_depth, out_alpha, out_percent_depth
		// DEBUG
		// ,dbg_ray_id, dbg_position, dbg_density, dbg_color, dbg_weight
	));

	return num_rendered;
}