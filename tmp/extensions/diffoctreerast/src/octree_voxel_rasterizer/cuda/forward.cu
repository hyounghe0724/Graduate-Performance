#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "config.h"
#include "auxiliary.h"
#include "data_structure.h"
#include "api.h"


/**
 * Helper function to find the highest bit set in an integer.
 * 
 * @param n Integer.
 * @return Highest bit set.
*/
static uint32_t getHigherMsb(uint32_t n) {
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}


/**
 * Forward pass for converting the input spherical harmonics coefficients of each voxel to a simple RGB color.
 * 
 * @param idx Index of the point in the input array.
 * @param deg Degree of the spherical harmonics coefficients.
 * @param max_coeffs Maximum number of coefficients.
 * @param pos Position of the point.
 * @param campos Camera position.
 * @param shs Array of spherical harmonics coefficients.
 * @param clamped Array of booleans to store if the color was clamped.
 * @return The color of the point.
 */
static __device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3 pos, glm::vec3 campos, const float* shs, bool* clamped) {
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0 | result.x > 1);
	clamped[3 * idx + 1] = (result.y < 0 | result.y > 1);
	clamped[3 * idx + 2] = (result.z < 0 | result.z > 1);
	return glm::min(glm::max(result, 0.0f), 1.0f);
}


/**
 * Compute the morton code for a 3D point based on the camera position and the depth of the voxel.
 * 
 * @param pos Position of the point.
 * @param campos Camera position.
 * @param depth Depth of the voxel.
 */
static __device__ uint32_t computeMortonCode(float3 pos, float3 campos, uint8_t depth) {
	uint32_t mul = 1 << MAX_TREE_DEPTH;
	uint32_t xcode = (uint32_t)(pos.x * mul);
	uint32_t ycode = (uint32_t)(pos.y * mul);
	uint32_t zcode = (uint32_t)(pos.z * mul);
	uint32_t cxcode = (uint32_t)(campos.x * mul);
	uint32_t cycode = (uint32_t)(campos.y * mul);
	uint32_t czcode = (uint32_t)(campos.z * mul);
	uint32_t xflip = 0, yflip = 0, zflip = 0;
	bool done = false;
	for (int i = 1; i <= MAX_TREE_DEPTH && !done; i++)
	{
		xflip |= ((xcode >> (MAX_TREE_DEPTH - i + 1) << 1) < (cxcode >> (MAX_TREE_DEPTH - i))) ? (1 << (MAX_TREE_DEPTH - i)) : 0;
		yflip |= ((ycode >> (MAX_TREE_DEPTH - i + 1) << 1) < (cycode >> (MAX_TREE_DEPTH - i))) ? (1 << (MAX_TREE_DEPTH - i)) : 0;
		zflip |= ((zcode >> (MAX_TREE_DEPTH - i + 1) << 1) < (czcode >> (MAX_TREE_DEPTH - i))) ? (1 << (MAX_TREE_DEPTH - i)) : 0;
		done = i == depth;
	}
	xcode ^= xflip;
	ycode ^= yflip;
	zcode ^= zflip;
	return expandBits(xcode) | (expandBits(ycode) << 1) | (expandBits(zcode) << 2);
}


/**
 * Preprocess input 3D points
 */
static __global__ void preprocess(
	const int num_nodes,
	const int active_sh_degree,
	const int num_sh_coefs,
	const float* positions,
	const uint8_t* tree_depths,
	const float scale_modifier,
	const float* shs,
	bool* clamped,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int width,
	const int height,
	const float tan_fovx,
	const float tan_fovy,
	const float focal_x,
	const float focal_y,
	const float* aabb,
	int4* bboxes,
	float* depths,
	float* rgb,
	const dim3 grid,
	uint32_t* tiles_touched,
	uint32_t* morton_codes
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= num_nodes)
		return;

	// Initialize bboxes and touched tiles to 0. If this isn't changed,
	// this voxel will not be processed further.
	bboxes[idx] = { 0, 0, 0, 0 };
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_orig = {
		positions[3 * idx] * aabb[3] + aabb[0],
		positions[3 * idx + 1] * aabb[4] + aabb[1],
		positions[3 * idx + 2] * aabb[5] + aabb[2]
	};
	float3 p_view;
	if (!in_frustum(idx, p_orig, viewmatrix, projmatrix, p_view))
		return;

	// Project 8 vertices of the voxel to screen space to find the
	// bounding box of the projected points.
	float nsize = powf(2.0f, -(float)tree_depths[idx]) * scale_modifier;
	float3 scale = { aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize };
	int4 bbox = get_bbox(p_orig, scale, projmatrix, width, height);
	uint2 rect_min, rect_max;
	getRect(bbox, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, active_sh_degree, num_sh_coefs, *(glm::vec3*)&p_orig, *cam_pos, shs, clamped);
		rgb[idx * 3 + 0] = result.x;
		rgb[idx * 3 + 1] = result.y;
		rgb[idx * 3 + 2] = result.z;
	}

	// Calculate view-dependent morton code for sorting.
	float3 pos = { positions[3 * idx], positions[3 * idx + 1], positions[3 * idx + 2] };
	float3 ncampos = {
		max(0.0f, min(1.0f, (cam_pos->x - aabb[0]) / aabb[3])),
		max(0.0f, min(1.0f, (cam_pos->y - aabb[1]) / aabb[4])),
		max(0.0f, min(1.0f, (cam_pos->z - aabb[2]) / aabb[5]))
	};
	uint32_t morton_code = computeMortonCode(pos, ncampos, tree_depths[idx]);

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	bboxes[idx] = bbox;
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
	morton_codes[idx] = morton_code;
}


/**
 * Generates one key/value pair for all voxel / tile overlaps. 
 * Run once per voxel (1:N mapping).
 * 
 * @param P Number of points.
 * @param points_xy 2D points.
 * @param depths Depths of points.
 * @param offsets Offsets for writing keys/values.
 * @param keys_unsorted Unsorted keys.
 * @param values_unsorted Unsorted values.
 * @param radii Radii of points.
 * @param grid Grid size.
 */
static __global__ void duplicateWithKeys(
	int P,
	const uint32_t* morton_codes,
	const uint32_t* offsets,
	uint64_t* keys_unsorted,
	uint32_t* values_unsorted,
	int4* bboxes,
	dim3 grid
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible voxels
	if (bboxes[idx].w > 0)
	{
		// Find this voxel's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;
		getRect(bboxes[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the voxel. Sorting the values 
		// with this key yields voxel IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= morton_codes[idx];
				keys_unsorted[off] = key;
				values_unsorted[off] = idx;
				off++;
			}
		}
	}
}


/**
 * Check keys to see if it is at the start/end of one tile's range in the full sorted list. If yes, write start/end of this tile.
 * 
 * @param L Number of points.
 * @param point_list_keys List of keys.
 * @param ranges Ranges of tiles.
 */
static __global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}


/**
 * Main rasterization method. Collaboratively works on one tile per
 * block, each thread treats one pixel. Alternates between fetching 
 * and rasterizing data.
 * 
 * @tparam CHANNELS Number of channels.
 * @param ranges Ranges of voxel instances for each tile.
 * @param point_list List of voxel instances.
 * @param W Width of the image.
 * @param H Height of the image.
 * @param bg_color Background color.
 * @param cam_pos Camera position.
 * @param tan_fovx Tangent of the horizontal field of view.
 * @param tan_fovy Tangent of the vertical field of view.
 * @param viewmatrix View matrix.
 * @param aabb Axis-aligned bounding box.
 * @param positions Centers of octree nodes.
 * @param features Features of octree nodes.
 * @param depths Depths of octree nodes (in view space).
 * @param tree_depths Depths of octree nodes.
 * @param scale_modifier Scale modifier.
 * @param densities densities of octree nodes.
 * @param final_T Final T.
 * @param final_wm_sum Weighted midpoint sum.
 * @param n_contrib Number of contributors.
 * @param out_color Output color.
 * @param out_depth Output depth.
 * @param out_alpha Output alpha.
 * @param out_distloss Output distance loss.
 */
template <uint32_t CHANNELS>
static __global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
render(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const int W,
	const int H,
	const float* __restrict__ bg_color,
	const float3* cam_pos,
	const float tan_fovx,
	const float tan_fovy,
	const float* __restrict__ viewmatrix,
	const float* __restrict__ aabb,
	const float* __restrict__ positions,
	const float* __restrict__ features,
	const float* __restrict__ depths,
	const uint8_t* __restrict__ tree_depths,
	const float scale_modifier,
	const float* __restrict__ densities,
	float* __restrict__ final_T,
	float* __restrict__ final_wm_sum,
	uint32_t* __restrict__ n_contrib,
	float* __restrict__ out_color,
	float* __restrict__ out_depth,
	float* __restrict__ out_alpha,
	float* __restrict__ out_distloss
) {
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;

	// Get ray direction and origin for this pixel.
	float3 ray_dir = getRayDir(pix, W, H, tan_fovx, tan_fovy, viewmatrix);

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float3 collected_xyz[BLOCK_SIZE];
	__shared__ float3 collected_scales[BLOCK_SIZE];
	__shared__ float collected_densities[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };
	float D = 0;
	float wm_prefix = 0;
	float distloss = 0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-voxel data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xyz[block.thread_rank()] = {
				positions[3 * coll_id] * aabb[3] + aabb[0],
				positions[3 * coll_id + 1] * aabb[4] + aabb[1],
				positions[3 * coll_id + 2] * aabb[5] + aabb[2]
			};
			float nsize = powf(2.0f, -(float)tree_depths[coll_id]) * scale_modifier;
			collected_scales[block.thread_rank()] = {aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize};
			collected_densities[block.thread_rank()] = densities[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Get ray-voxel intersection
			float3 p = collected_xyz[j];
			float3 scale = collected_scales[j];
			float3 voxel_min = { p.x - 0.5f * scale.x, p.y - 0.5f * scale.y, p.z - 0.5f * scale.z };
			float3 voxel_max = { p.x + 0.5f * scale.x, p.y + 0.5f * scale.y, p.z + 0.5f * scale.z };
			float2 itsc = get_ray_voxel_intersection(*cam_pos, ray_dir, voxel_min, voxel_max);
			float itsc_dist = (itsc.y >= itsc.x) ? itsc.y - itsc.x : -1.0f;
			if (itsc_dist <= 0.0f)
				continue;

			// Volume rendering
			float alpha = min(1 - exp(-collected_densities[j] * itsc_dist), 0.999f);
			const float weight = alpha * T;

			// Accumulate color and depth
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * weight;
			D += depths[collected_id[j]] * weight;

			// Distortion loss
			// loss_bi := 2 * (wm * w_prefix - w * wm_prefix); loss_uni := 1.0f / 3.0f * (itsc_dist * w^2);
			if (out_distloss != nullptr)
			{
				float midpoint = 0.5f * (itsc.x + itsc.y);
				float wm = weight * midpoint;
				distloss += 2.0f * (wm * (1.0f - T) - weight * wm_prefix) + (1.0f / 3.0f) * itsc_dist * weight * weight;
				wm_prefix += wm;
			}

			T *= 1 - alpha;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;

			// If we have accumulated enough, we can stop
			if (T < 0.001f)
				done = true;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		out_depth[pix_id] = D;
		out_alpha[pix_id] = 1.0f - T;
		if (out_distloss != nullptr) {
			out_distloss[pix_id] = distloss;
			final_wm_sum[pix_id] = wm_prefix;
		}
	}
}

int OctreeVoxelRasterizer::CUDA::forward(
	std::function<char*(size_t)> geometryBuffer,
	std::function<char*(size_t)> binningBuffer,
	std::function<char*(size_t)> imageBuffer,
	const int num_nodes,
	const int active_sh_degree,
	const int num_sh_coefs,
	const float* background,
    const int width,
    const int height,
    const float* aabb,
    const float* positions,
    const float* shs,
    const float* colors_precomp,
    const float* densities,
    const uint8_t* depths,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
    const float* cam_pos,
	const float tan_fovx,
    const float tan_fovy,
    float* out_color,
    float* out_depth,
    float* out_alpha,
	float* out_distloss
) {
	DEBUG_PRINT("Starting forward pass\n");
	DEBUG_PRINT("    - Number of nodes: %d\n", num_nodes);
	DEBUG_PRINT("    - Active SH degree: %d\n", active_sh_degree);
	DEBUG_PRINT("    - Number of SH coefficients: %d\n", num_sh_coefs);
	DEBUG_PRINT("    - Image size: %d x %d\n", width, height);

	// Parrallel config (2D grid of 2D blocks)
	dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Allocate buffers for auxiliary info for points and pixels
	DEBUG_PRINT("Allocating buffers\n");
	size_t buffer_size;
	char* buffer_ptr;
	buffer_size = required<GeometryState>(num_nodes);
	DEBUG_PRINT("    - Geometry buffer size: %zu\n", buffer_size);
	buffer_ptr = geometryBuffer(buffer_size);
	GeometryState geomState = GeometryState::fromChunk(buffer_ptr, num_nodes);
	buffer_size = required<ImageState>(width * height);
	DEBUG_PRINT("    - Image buffer size: %zu\n", buffer_size);
	buffer_ptr = imageBuffer(buffer_size);
	ImageState imgState = ImageState::fromChunk(buffer_ptr, width * height);

	const float focal_x = height / (2.f * tan_fovy);
	const float focal_y = width / (2.f * tan_fovx);

	// Run preprocessing kernel
	DEBUG_PRINT("Calling preprocess kernel\n");
	CHECK_CUDA(preprocess<<<(num_nodes+255)/256, 256>>>(
		num_nodes, active_sh_degree, num_sh_coefs,
		positions, depths, scale_modifier,
		shs, geomState.clamped, colors_precomp,
		viewmatrix, projmatrix, (glm::vec3*)cam_pos,
		width, height, tan_fovx, tan_fovy, focal_x, focal_y, aabb,
		geomState.bboxes, geomState.depths, geomState.rgb,
		grid, geomState.tiles_touched, geomState.morton_codes
	));

	// Compute prefix sum over full list of touched tile counts by voxels
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(
		geomState.scanning_space, geomState.scan_size,
		geomState.tiles_touched, geomState.point_offsets, num_nodes
	));

	// Retrieve total number of voxel instances to launch
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + num_nodes - 1, sizeof(int), hipMemcpyDeviceToHost));
	if (num_rendered == 0)
		return 0;

	// Allocate buffer for binning state
	DEBUG_PRINT("Allocating binning buffer\n");
	DEBUG_PRINT("    - Number of rendered nodes: %d\n", num_rendered);
	buffer_size = required<BinningState>(num_rendered);
	DEBUG_PRINT("    - Binning buffer size: %zu\n", buffer_size);
	buffer_ptr = binningBuffer(buffer_size);
	BinningState binningState = BinningState::fromChunk(buffer_ptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated voxel indices to be sorted
	DEBUG_PRINT("Calling duplicateWithKeys kernel\n");
	CHECK_CUDA(duplicateWithKeys<<<(num_nodes+255)/256, 256>>>(
		num_nodes, geomState.morton_codes, geomState.point_offsets,
		binningState.point_list_keys_unsorted, binningState.point_list_unsorted,
		geomState.bboxes, grid
	));

	// Sort complete list of (duplicated) voxel indices by keys
	int bit = getHigherMsb(grid.x * grid.y);
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space, binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit
	));

	// Identify start and end of per-tile workloads in sorted list
	CHECK_CUDA(hipMemset(imgState.ranges, 0, grid.x * grid.y * sizeof(uint2)));
	CHECK_CUDA(identifyTileRanges<<<(num_rendered+255)/256, 256>>>(
		num_rendered, binningState.point_list_keys, imgState.ranges
	));

	// Let each tile blend its range of voxels independently in parallel
	const float* color_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	DEBUG_PRINT("Calling render kernel\n");
	CHECK_CUDA(render<NUM_CHANNELS><<<grid, block>>>(
		imgState.ranges, binningState.point_list,
		width, height, background,
		(float3*)cam_pos, tan_fovx, tan_fovy, viewmatrix, aabb,
		positions, color_ptr, geomState.depths, depths, scale_modifier, densities,
		imgState.accum_alpha, imgState.wm_sum, imgState.n_contrib,
		out_color, out_depth, out_alpha, out_distloss
	));

	return num_rendered;
}