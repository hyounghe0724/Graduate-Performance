#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <glm/glm.hpp>
namespace cg = cooperative_groups;

#include "config.h"
#include "auxiliary.h"
#include "data_structure.h"
#include "api.h"


/**
 * Backward pass for converting the input spherical harmonics coefficients of each voxel to a simple RGB color.
 * 
 * @param deg Degree of the spherical harmonics coefficients.
 * @param max_coeffs Maximum number of coefficients.
 * @param mean Array of 3D points.
 * @param campos Camera position.
 * @param sh Array of spherical harmonics coefficients.
 * @param dL_dcolor Gradient of the output colors.
 * @param dL_dsh Gradient of the input spherical harmonics coefficients.
 */
static __device__ void computeColorFromSHBackward(int deg, int max_coeffs, const glm::vec3* mean, glm::vec3 campos, const float* sh, const float* dL_dcolor, float* dL_dsh) {
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = *mean;
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	for (int tc = 0; tc < DECOUPOLY_RANK; tc++)
		for (int ch = 0; ch < CHANNELS; ch++)
			dL_dsh[tc * max_coeffs * CHANNELS + ch] = SH_C0 * dL_dcolor[tc * CHANNELS + ch];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		float coeff[3] = { -SH_C1 * y, SH_C1 * z, -SH_C1 * x };
		for (int tc = 0; tc < DECOUPOLY_RANK; tc++) {
			for (int ch = 0; ch < CHANNELS; ch++) {
				dL_dsh[tc * max_coeffs * CHANNELS + 1 * CHANNELS + ch] = coeff[0] * dL_dcolor[tc * CHANNELS + ch];
				dL_dsh[tc * max_coeffs * CHANNELS + 2 * CHANNELS + ch] = coeff[1] * dL_dcolor[tc * CHANNELS + ch];
				dL_dsh[tc * max_coeffs * CHANNELS + 3 * CHANNELS + ch] = coeff[2] * dL_dcolor[tc * CHANNELS + ch];
			}
		}

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			float coeff[5] = { SH_C2[0] * xy, SH_C2[1] * yz, SH_C2[2] * (2.0f * zz - xx - yy), SH_C2[3] * xz, SH_C2[4] * (xx - yy) };
			for (int tc = 0; tc < DECOUPOLY_RANK; tc++) {
				for (int ch = 0; ch < CHANNELS; ch++) {
					dL_dsh[tc * max_coeffs * CHANNELS + 4 * CHANNELS + ch] = coeff[0] * dL_dcolor[tc * CHANNELS + ch];
					dL_dsh[tc * max_coeffs * CHANNELS + 5 * CHANNELS + ch] = coeff[1] * dL_dcolor[tc * CHANNELS + ch];
					dL_dsh[tc * max_coeffs * CHANNELS + 6 * CHANNELS + ch] = coeff[2] * dL_dcolor[tc * CHANNELS + ch];
					dL_dsh[tc * max_coeffs * CHANNELS + 7 * CHANNELS + ch] = coeff[3] * dL_dcolor[tc * CHANNELS + ch];
					dL_dsh[tc * max_coeffs * CHANNELS + 8 * CHANNELS + ch] = coeff[4] * dL_dcolor[tc * CHANNELS + ch];
				}
			}

			if (deg > 2)
			{
				float coeff[7] = {
					SH_C3[0] * y * (3.0f * xx - yy),
					SH_C3[1] * xy * z,
					SH_C3[2] * y * (4.0f * zz - xx - yy),
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy),
					SH_C3[4] * x * (4.0f * zz - xx - yy),
					SH_C3[5] * z * (xx - yy),
					SH_C3[6] * x * (xx - 3.0f * yy)
				};
				for (int tc = 0; tc < DECOUPOLY_RANK; tc++) {
					for (int ch = 0; ch < CHANNELS; ch++) {
						dL_dsh[tc * max_coeffs * CHANNELS + 9 * CHANNELS + ch] = coeff[0] * dL_dcolor[tc * CHANNELS + ch];
						dL_dsh[tc * max_coeffs * CHANNELS + 10 * CHANNELS + ch] = coeff[1] * dL_dcolor[tc * CHANNELS + ch];
						dL_dsh[tc * max_coeffs * CHANNELS + 11 * CHANNELS + ch] = coeff[2] * dL_dcolor[tc * CHANNELS + ch];
						dL_dsh[tc * max_coeffs * CHANNELS + 12 * CHANNELS + ch] = coeff[3] * dL_dcolor[tc * CHANNELS + ch];
						dL_dsh[tc * max_coeffs * CHANNELS + 13 * CHANNELS + ch] = coeff[4] * dL_dcolor[tc * CHANNELS + ch];
						dL_dsh[tc * max_coeffs * CHANNELS + 14 * CHANNELS + ch] = coeff[5] * dL_dcolor[tc * CHANNELS + ch];
						dL_dsh[tc * max_coeffs * CHANNELS + 15 * CHANNELS + ch] = coeff[6] * dL_dcolor[tc * CHANNELS + ch];
					}
				}
			}
		}
	}
}


/**
 * Backward pass of the preprocessing steps
 */
static __global__ void preprocessBackward(
	const int num_points,
	const int active_sh_degree,
	const int num_sh_coefs,
	const float* positions,
	const float* shs,
	const glm::vec3* cam_pos,
	const float* aabb,
    float* grad_colors,
    float* grad_shs
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= num_points)
		return;

	float3 p_orig = {
		positions[3 * idx] * aabb[3] + aabb[0],
		positions[3 * idx + 1] * aabb[4] + aabb[1],
		positions[3 * idx + 2] * aabb[5] + aabb[2]
	};

	// Compute gradient updates due to computing colors from SHs
	if (shs)
		computeColorFromSHBackward(
            active_sh_degree, num_sh_coefs, (glm::vec3*)&p_orig, *cam_pos,
            shs + idx * DECOUPOLY_RANK * num_sh_coefs * CHANNELS,
			grad_colors + idx * DECOUPOLY_RANK * CHANNELS,
			grad_shs + idx * DECOUPOLY_RANK * num_sh_coefs * CHANNELS
       	);
}


static __device__ void sample_decoupoly(
	const float* decoupoly_V,
	const float* decoupoly_g,
	const float* densities,
	const float* colors,
	const float density_shift,
	const int used_rank,
	const float3& p,
	const float3& voxel_min,
	const float3& voxel_max,
	float& out_density,
	float* out_color
) {
	float3 _p = {
		((p.x - voxel_min.x) / (voxel_max.x - voxel_min.x) - 0.5f) * 2.0f,
		((p.y - voxel_min.y) / (voxel_max.y - voxel_min.y) - 0.5f) * 2.0f,
		((p.z - voxel_min.z) / (voxel_max.z - voxel_min.z) - 0.5f) * 2.0f
	};

	float _value;
	float _x, _x_n;
	for (int i = 0; i < used_rank; i++) {
		_x_n = 1.0f;
		_x = decoupoly_V[i * 3 + 0] * _p.x + decoupoly_V[i * 3 + 1] * _p.y + decoupoly_V[i * 3 + 2] * _p.z;
		_value = 1.0f;
		for (int j = 0; j < DECOUPOLY_DEGREE; j++) {
			_value += decoupoly_g[i * DECOUPOLY_DEGREE + j] * _x_n;
			_x_n *= _x;
		}

		out_density += densities[i] * _value;
		for (int j = 0; j < CHANNELS; j++)
			out_color[j] += colors[CHANNELS * i + j] * _value;
	}
	for (int j = 0; j < CHANNELS; j++)
		out_color[j] = sigmoid(out_color[j]);
	out_density = softplus(out_density - density_shift * 10) * min(1 / (1 - density_shift), 25.0f);
}


static __device__ void sample_decoupoly_backward(
	const float* decoupoly_V,
	const float* decoupoly_g,
	const float* densities,
	const float* colors,
	const float density_shift,
	const int used_rank,
	const float& density,
	const float* color,
	const float& grad_density,
	const float* grad_color,
	const float3& p,
	const float3& voxel_min,
	const float3& voxel_max,
	float* decoupoly_V_grad,
	float* decoupoly_g_grad,
	float* density_grad,
	float* color_grad
) {
	float3 _p = {
		((p.x - voxel_min.x) / (voxel_max.x - voxel_min.x) - 0.5f) * 2.0f,
		((p.y - voxel_min.y) / (voxel_max.y - voxel_min.y) - 0.5f) * 2.0f,
		((p.z - voxel_min.z) / (voxel_max.z - voxel_min.z) - 0.5f) * 2.0f
	};

	float ramp = min(1 / (1 - density_shift), 25.0f);
	float _grad_density = grad_density * softplus_prime(density / ramp) * ramp;
	float _grad_color[CHANNELS];
	for (int j = 0; j < CHANNELS; j++)
		_grad_color[j] = grad_color[j] * sigmoid_prime(color[j]);

	float _value;
	float _x, _x_n, _grad_value, _grad_x;
	for (int i = 0; i < used_rank; i++) {
		_x_n = 1.0f;
		_x = decoupoly_V[i * 3 + 0] * _p.x + decoupoly_V[i * 3 + 1] * _p.y + decoupoly_V[i * 3 + 2] * _p.z;
		_value = 1.0f;

		// grad of _value
		_grad_value = _grad_density * densities[i];
		for (int j = 0; j < CHANNELS; j++)
			_grad_value += _grad_color[j] * colors[CHANNELS * i + j];

		_grad_x = 0;
		for (int j = 0; j < DECOUPOLY_DEGREE; j++) {
			_value += decoupoly_g[i * DECOUPOLY_DEGREE + j] * _x_n;

			// grad of _x
			if (j < DECOUPOLY_DEGREE - 1)
				_grad_x += decoupoly_g[i * DECOUPOLY_DEGREE + j + 1] * _grad_value * _x_n * (j + 1);

			// grad of decoupoly_g
			atomicAdd(decoupoly_g_grad + i * DECOUPOLY_DEGREE + j, _grad_value * _x_n);

			_x_n *= _x;
		}

		// grad of decoupoly_V
		atomicAdd(decoupoly_V_grad + i * 3 + 0, _grad_x * _p.x);
		atomicAdd(decoupoly_V_grad + i * 3 + 1, _grad_x * _p.y);
		atomicAdd(decoupoly_V_grad + i * 3 + 2, _grad_x * _p.z);
		
		// grad of density
		atomicAdd(density_grad + i, _grad_density * _value);
		
		// grad of color
		for (int j = 0; j < CHANNELS; j++) {
			atomicAdd(color_grad + i * CHANNELS + j, _grad_color[j] * _value);
		}
	}
}


static __device__ void sample_decoupoly_backward_local(
	const float* decoupoly_V,
	const float* decoupoly_g,
	const float* densities,
	const float* colors,
	const float density_shift,
	const int used_rank,
	const float& density,
	const float* color,
	const float& grad_density,
	const float* grad_color,
	const float3& p,
	const float3& voxel_min,
	const float3& voxel_max,
	float* decoupoly_V_grad,
	float* decoupoly_g_grad,
	float* density_grad,
	float* color_grad
) {
	float3 _p = {
		((p.x - voxel_min.x) / (voxel_max.x - voxel_min.x) - 0.5f) * 2.0f,
		((p.y - voxel_min.y) / (voxel_max.y - voxel_min.y) - 0.5f) * 2.0f,
		((p.z - voxel_min.z) / (voxel_max.z - voxel_min.z) - 0.5f) * 2.0f
	};

	float ramp = min(1 / (1 - density_shift), 25.0f);
	float _grad_density = grad_density * softplus_prime(density / ramp) * ramp;
	float _grad_color[CHANNELS];
	for (int j = 0; j < CHANNELS; j++)
		_grad_color[j] = grad_color[j] * sigmoid_prime(color[j]);

	float _value;
	float _x, _x_n, _grad_value, _grad_x;
	for (int i = 0; i < used_rank; i++) {
		_x_n = 1.0f;
		_x = decoupoly_V[i * 3 + 0] * _p.x + decoupoly_V[i * 3 + 1] * _p.y + decoupoly_V[i * 3 + 2] * _p.z;
		_value = 1.0f;

		// grad of _value
		_grad_value = _grad_density * densities[i];
		for (int j = 0; j < CHANNELS; j++)
			_grad_value += _grad_color[j] * colors[CHANNELS * i + j];

		_grad_x = 0;
		for (int j = 0; j < DECOUPOLY_DEGREE; j++) {
			_value += decoupoly_g[i * DECOUPOLY_DEGREE + j] * _x_n;

			// grad of _x
			if (j < DECOUPOLY_DEGREE - 1)
				_grad_x += decoupoly_g[i * DECOUPOLY_DEGREE + j + 1] * _grad_value * _x_n * (j + 1);

			// grad of decoupoly_g
			decoupoly_g_grad[i * DECOUPOLY_DEGREE + j] += _grad_value * _x_n;

			_x_n *= _x;
		}

		// grad of decoupoly_V
		decoupoly_V_grad[i * 3 + 0] += _grad_x * _p.x;
		decoupoly_V_grad[i * 3 + 1] += _grad_x * _p.y;
		decoupoly_V_grad[i * 3 + 2] += _grad_x * _p.z;
		
		// grad of density
		density_grad[i] += _grad_density * _value;
		
		// grad of color
		for (int j = 0; j < CHANNELS; j++) {
			color_grad[i * CHANNELS + j] += _grad_color[j] * _value;
		}
	}
}


/**
 * Backward version of the rendering procedure.
 * 
 * @param ranges Ranges of voxel instances for each tile.
 * @param point_list List of voxel instances.
 * @param W Width of the image.
 * @param H Height of the image.
 * @param bg_color Background color.
 * @param cam_pos Camera position.
 * @param tan_fovx Tangent of the horizontal field of view.
 * @param tan_fovy Tangent of the vertical field of view.
 * @param viewmatrix View matrix.
 * @param aabb Axis-aligned bounding box.
 * @param positions Positions of octree nodes.
 * @param decoupolys Decoupoly features of octree nodes.
 * @param densities Densities of octree nodes.
 * @param density_shift Shift of densities.
 * @param colors Colors of octree nodes.
 * @param used_rank Rank of the used decoupoly channel.
 * @param tree_depths Depths of octree nodes.
 * @param scale_modifier Scale modifier.
 * @param random_image Random image.
 * @param densities densities of octree nodes.
 * @param n_contrib Number of contributors.
 * @param out_color Output color.
 * @param out_depth Output depth.
 * @param out_alpha Output alpha.
 * @param grad_out_colors Gradient of output colors.
 * @param grad_out_depths Gradient of output depths.
 * @param grad_out_alphas Gradient of output alphas.
 * @param grad_decoupolys_V Gradient of decoupolys.
 * @param grad_decoupolys_g Gradient of decoupolys.
 * @param grad_densities Gradient of densities.
 * @param grad_colors Gradient of colors.
 * @param aux_grad_colors2 Auxiliary gradient of squared colors.
 * @param aux_contributions Auxiliary contributions.
 */
static __global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderBackward(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const int W, 
    const int H,
	const float* __restrict__ bg_color,
	const float3* cam_pos,
	const float tan_fovx,
	const float tan_fovy,
	const float* __restrict__ viewmatrix,
	const float* __restrict__ aabb,
	const float* __restrict__ positions,
	const float* __restrict__ decoupolys_V,
	const float* __restrict__ decoupolys_g,
	const float* __restrict__ densities,
	const float density_shift,
	const float* __restrict__ colors,
	const int used_rank,
	const uint8_t* __restrict__ tree_depths,
	const float scale_modifier,
	const float* __restrict__ random_image,
	const uint32_t* __restrict__ n_contrib,
	const uint32_t* __restrict__ t_contrib,
	const float* __restrict__ out_color,
	const float* __restrict__ out_depth,
	const float* __restrict__ out_alpha,
	const float* __restrict__ grad_out_colors,
	const float* __restrict__ grad_out_depths,
	const float* __restrict__ grad_out_alphas,
    float* __restrict__ grad_decoupolys_V,
	float* __restrict__ grad_decoupolys_g,
	float* __restrict__ grad_densities,
	float* __restrict__ grad_colors,
    float* __restrict__ aux_grad_colors2,
    float* __restrict__ aux_contributions
) {
    // We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	
	// Get ray direction and origin for this pixel.
	float3 ray_dir = getRayDir(pix, W, H, tan_fovx, tan_fovy, viewmatrix);

    const bool inside = pix.x < W&& pix.y < H;
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + PREFETCH_BUFFER_SIZE - 1) / PREFETCH_BUFFER_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for shared memory
	__shared__ uint32_t collected_ids[PREFETCH_BUFFER_SIZE];
	__shared__ float3 collected_scales[PREFETCH_BUFFER_SIZE];
	__shared__ float3 collected_xyz[PREFETCH_BUFFER_SIZE];
	__shared__ float collected_decoupolys_V[PREFETCH_BUFFER_SIZE * DECOUPOLY_V_SIZE];
	__shared__ float collected_decoupolys_g[PREFETCH_BUFFER_SIZE * DECOUPOLY_G_SIZE];
	__shared__ float collected_densities[PREFETCH_BUFFER_SIZE * DECOUPOLY_RANK];
	__shared__ float collected_colors[PREFETCH_BUFFER_SIZE * DECOUPOLY_RANK * CHANNELS];
	#ifdef GRAD_SHARED_TO_GLOBAL
	__shared__ float grad_collected_decoupolys_V[PREFETCH_BUFFER_SIZE * DECOUPOLY_V_SIZE];
	for (int i = block.thread_rank(); i < PREFETCH_BUFFER_SIZE * DECOUPOLY_V_SIZE; i += BLOCK_SIZE)
		grad_collected_decoupolys_V[i] = 0;
	__shared__ float grad_collected_decoupolys_g[PREFETCH_BUFFER_SIZE * DECOUPOLY_G_SIZE];
	for (int i = block.thread_rank(); i < PREFETCH_BUFFER_SIZE * DECOUPOLY_G_SIZE; i += BLOCK_SIZE)
		grad_collected_decoupolys_g[i] = 0;
	__shared__ float grad_collected_densities[PREFETCH_BUFFER_SIZE * DECOUPOLY_RANK];
	for (int i = block.thread_rank(); i < PREFETCH_BUFFER_SIZE * DECOUPOLY_RANK; i += BLOCK_SIZE)
		grad_collected_densities[i] = 0;
	__shared__ float grad_collected_colors[PREFETCH_BUFFER_SIZE * DECOUPOLY_RANK * CHANNELS];
	for (int i = block.thread_rank(); i < PREFETCH_BUFFER_SIZE * DECOUPOLY_RANK * CHANNELS; i += BLOCK_SIZE)
		grad_collected_colors[i] = 0;
	#elif defined(GRAD_LOCAL_REDUCED_TO_GLOBAL)
		typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
		__shared__ typename BlockReduce::TempStorage temp_storage;
		__shared__ float shared_grad_decoupolys_V[DECOUPOLY_V_SIZE];
		__shared__ float shared_grad_decoupolys_g[DECOUPOLY_G_SIZE];
		__shared__ float shared_grad_densities[DECOUPOLY_RANK];
		__shared__ float shared_grad_colors[DECOUPOLY_RANK * CHANNELS];
	#endif

    // Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = inside ? n_contrib[pix_id] : 0;
	uint32_t last_t = inside ? t_contrib[pix_id] : 0;
	float C[CHANNELS] = { 0 };
	float D = 0;
	float final_T = 1.0f - out_alpha[pix_id];
	float final_C[CHANNELS];
	for (int i = 0; i < CHANNELS; i++)
		final_C[i] = out_color[i * H * W + pix_id];
	float final_D = out_depth[pix_id];
	float jitter = random_image[pix_id];
	float dL_dout_color[CHANNELS];
	float dL_dout_depth;
	float dL_dout_alpha;
	if (inside) {
		if (grad_out_colors != nullptr) {
			for (int i = 0; i < CHANNELS; i++)
				dL_dout_color[i] = grad_out_colors[i * H * W + pix_id];
		}
		if (grad_out_depths != nullptr)
			dL_dout_depth = grad_out_depths[pix_id];
		if (grad_out_alphas != nullptr)
			dL_dout_alpha = grad_out_alphas[pix_id];
	}

	// Traverse all voxels
	for (int i = 0; i < rounds; i++, toDo -= PREFETCH_BUFFER_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-voxel data from global to shared
		#ifndef ASYNC_GLOBAL_TO_SHARED
		for (int j = 0; j < PREFETCH_BUFFER_SIZE; j++)
		{
			int progress = i * PREFETCH_BUFFER_SIZE + j;
			if (range.x + progress < range.y)
			{
				int coll_id = point_list[range.x + progress];
				float nsize = powf(2.0f, -(float)tree_depths[coll_id]) * scale_modifier;
				if (block.thread_rank() == 0) {
					collected_scales[j] = { aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize };
					collected_xyz[j] = {
						positions[3 * coll_id] * aabb[3] + aabb[0],
						positions[3 * coll_id + 1] * aabb[4] + aabb[1],
						positions[3 * coll_id + 2] * aabb[5] + aabb[2]
					};
				}
				for (int ch = block.thread_rank(); ch < DECOUPOLY_V_SIZE; ch += BLOCK_SIZE)
					collected_decoupolys_V[j * DECOUPOLY_V_SIZE + ch] = decoupolys_V[coll_id * DECOUPOLY_V_SIZE + ch];
				for (int ch = block.thread_rank(); ch < DECOUPOLY_G_SIZE; ch += BLOCK_SIZE)
					collected_decoupolys_g[j * DECOUPOLY_G_SIZE + ch] = decoupolys_g[coll_id * DECOUPOLY_G_SIZE + ch];
				for (int ch = block.thread_rank(); ch < DECOUPOLY_RANK; ch += BLOCK_SIZE)
					collected_densities[j * DECOUPOLY_RANK + ch] = densities[coll_id * DECOUPOLY_RANK + ch];
				for (int ch = block.thread_rank(); ch < DECOUPOLY_RANK * CHANNELS; ch += BLOCK_SIZE)
					collected_colors[j * DECOUPOLY_RANK * CHANNELS + ch] = colors[coll_id * DECOUPOLY_RANK * CHANNELS + ch];
			}
		}
		block.sync();
		#else
		for (int j = 0; j < PREFETCH_BUFFER_SIZE; j++)
		{
			int progress = i * PREFETCH_BUFFER_SIZE + j;
			if (range.x + progress < range.y)
			{
				int coll_id = point_list[range.x + progress];
				float nsize = powf(2.0f, -(float)tree_depths[coll_id]) * scale_modifier;
				if (block.thread_rank() == 0) {
					collected_ids[j] = coll_id;
					collected_scales[j] = { aabb[3] * nsize, aabb[4] * nsize, aabb[5] * nsize };
					collected_xyz[j] = {
						positions[3 * coll_id] * aabb[3] + aabb[0],
						positions[3 * coll_id + 1] * aabb[4] + aabb[1],
						positions[3 * coll_id + 2] * aabb[5] + aabb[2]
					};
				}
				cg::memcpy_async(block, collected_decoupolys_V + j * DECOUPOLY_V_SIZE, decoupolys_V + coll_id * DECOUPOLY_V_SIZE, DECOUPOLY_V_SIZE * sizeof(float));
				cg::memcpy_async(block, collected_decoupolys_g + j * DECOUPOLY_G_SIZE, decoupolys_g + coll_id * DECOUPOLY_G_SIZE, DECOUPOLY_G_SIZE * sizeof(float));
				cg::memcpy_async(block, collected_densities + j * DECOUPOLY_RANK, densities + coll_id * DECOUPOLY_RANK, DECOUPOLY_RANK * sizeof(float));
				cg::memcpy_async(block, collected_colors + j * DECOUPOLY_RANK * CHANNELS, colors + coll_id * DECOUPOLY_RANK * CHANNELS, DECOUPOLY_RANK * CHANNELS * sizeof(float));
			}
		}
		cg::wait(block);
		block.sync();
		#endif	

		// Iterate over current batch
		for (int j = 0; j < min(PREFETCH_BUFFER_SIZE, toDo); j++)
		{
			contributor++;
			if (contributor > last_contributor)
				done = true;

			#ifdef GRAD_LOCAL_REDUCED_TO_GLOBAL
			// End if entire block votes that it is done rasterizing
			int num_done = __syncthreads_count(done);
			if (num_done == BLOCK_SIZE)
				break;
			#else
			// End if each thread is done
			if (done)
				break;
			#endif

			// Get ray-voxel intersection
			float3 p = collected_xyz[j];
			float3 scale = collected_scales[j];
			float3 voxel_min = { p.x - 0.5f * scale.x, p.y - 0.5f * scale.y, p.z - 0.5f * scale.z };
			float3 voxel_max = { p.x + 0.5f * scale.x, p.y + 0.5f * scale.y, p.z + 0.5f * scale.z };
			float2 itsc = get_ray_voxel_intersection(*cam_pos, ray_dir, voxel_min, voxel_max);

			// Ray marching
			#if defined(GRAD_LOCAL_TO_GLOBAL) || defined(GRAD_LOCAL_REDUCED_TO_GLOBAL)
			float local_grad_decoupolys_V[DECOUPOLY_V_SIZE] = { 0 };
			float local_grad_decoupolys_g[DECOUPOLY_G_SIZE] = { 0 };
			float local_grad_densities[DECOUPOLY_RANK] = { 0 };
			float local_grad_colors[DECOUPOLY_RANK * CHANNELS] = { 0 };
			#endif
			float step = (0.5f / SAMPLE_DIM) * scale.x;
			int t_start = (int)ceil(itsc.x / step - jitter);
			int t_end = (int)floor(itsc.y / step - jitter);
			float w_sum = 0.0f;
			for (int t = t_start; t <= t_end; t++) {
				if (t > last_t)
					break;

				float z = (t + jitter) * step;

				// Sample decoupoly
				float density = 0;
				float color[CHANNELS] = { 0 };
				p = { cam_pos->x + z * ray_dir.x, cam_pos->y + z * ray_dir.y, cam_pos->z + z * ray_dir.z };
				sample_decoupoly(
					collected_decoupolys_V + j * DECOUPOLY_V_SIZE,
					collected_decoupolys_g + j * DECOUPOLY_G_SIZE,
					collected_densities + j * DECOUPOLY_RANK,
					collected_colors + j * DECOUPOLY_RANK * CHANNELS,
					density_shift,
					used_rank,
					p, voxel_min, voxel_max,
					density, color
				);

				// Accumulate
				float alpha = min(1 - exp(-density * step), 0.999f);
				const float weight = alpha * T;
				w_sum += weight;
				for (int k = 0; k < CHANNELS; k++)
					C[k] += color[k] * weight;
				D += z * weight;

				T *= 1 - alpha;

				// Residual
				float residual_T = final_T / T;
				float residual_C[CHANNELS];
				for (int i = 0; i < CHANNELS; i++)
					residual_C[i] = (final_C[i] - C[i]) / T;
				float residual_D = (final_D - D) / T;

				// Propagate gradients
				float dL_dalpha = 0.0f;
				float dL_dcolor[CHANNELS] = { 0 };
				// color
				if (grad_out_colors != nullptr) {
					for (int i = 0; i < CHANNELS; i++) {
						dL_dalpha += (color[i] - residual_C[i]) * dL_dout_color[i];
						dL_dcolor[i] = weight * dL_dout_color[i];
					}
				}
				// depth
				if (grad_out_depths != nullptr)
					dL_dalpha += (z - residual_D) * dL_dout_depth;
				// alpha
				if (grad_out_alphas != nullptr)
					dL_dalpha += residual_T * dL_dout_alpha;
				dL_dalpha *= T / (1.0f - alpha);
				float dL_ddensity = dL_dalpha * step * (1 - alpha);

				// Propagate gradients to trivec
				#if defined(GRAD_GLOBAL)
				sample_decoupoly_backward(
					collected_decoupolys_V + j * DECOUPOLY_V_SIZE,
					collected_decoupolys_g + j * DECOUPOLY_G_SIZE,
					collected_densities + j * DECOUPOLY_RANK,
					collected_colors + j * DECOUPOLY_RANK * CHANNELS,
					density_shift,
					used_rank,
					density, color,
					dL_ddensity, dL_dcolor,
					p, voxel_min, voxel_max,
					grad_decoupolys_V + collected_ids[j] * DECOUPOLY_V_SIZE,
					grad_decoupolys_g + collected_ids[j] * DECOUPOLY_G_SIZE,
					grad_densities + collected_ids[j] * DECOUPOLY_RANK,
					grad_colors + collected_ids[j] * DECOUPOLY_RANK * CHANNELS
				);
				#elif defined(GRAD_SHARED_TO_GLOBAL)
				sample_decoupoly_backward(
					collected_decoupolys_V + j * DECOUPOLY_V_SIZE,
					collected_decoupolys_g + j * DECOUPOLY_G_SIZE,
					collected_densities + j * DECOUPOLY_RANK,
					collected_colors + j * DECOUPOLY_RANK * CHANNELS,
					density_shift,
					used_rank,
					density, color,
					dL_ddensity, dL_dcolor,
					p, voxel_min, voxel_max,
					grad_collected_decoupolys_V + j * DECOUPOLY_V_SIZE,
					grad_collected_decoupolys_g + j * DECOUPOLY_G_SIZE,
					grad_collected_densities + j * DECOUPOLY_RANK,
					grad_collected_colors + j * DECOUPOLY_RANK * CHANNELS
				);
				#elif defined(GRAD_LOCAL_TO_GLOBAL) || defined(GRAD_LOCAL_REDUCED_TO_GLOBAL)
				sample_decoupoly_backward_local(
					collected_decoupolys_V + j * DECOUPOLY_V_SIZE,
					collected_decoupolys_g + j * DECOUPOLY_G_SIZE,
					collected_densities + j * DECOUPOLY_RANK,
					collected_colors + j * DECOUPOLY_RANK * CHANNELS,
					density_shift,
					used_rank,
					density, color,
					dL_ddensity, dL_dcolor,
					p, voxel_min, voxel_max,
					local_grad_decoupolys_V, local_grad_decoupolys_g, local_grad_densities, local_grad_colors
				);
				#endif

				if (T < 0.001f)
					break;
			}

			// Copy gradients to global memory
			#if defined(GRAD_LOCAL_TO_GLOBAL)
			for (int ch = 0; ch < DECOUPOLY_V_SIZE; ch++) {
				// shuffle the atomic adds to avoid conflicts
				int _ch = (ch + block.thread_rank()) % DECOUPOLY_V_SIZE;
				atomicAdd(grad_decoupolys_V + collected_ids[j] * DECOUPOLY_V_SIZE + _ch, local_grad_decoupolys_V[_ch]);
			}
			for (int ch = 0; ch < DECOUPOLY_G_SIZE; ch++) {
				// shuffle the atomic adds to avoid conflicts
				int _ch = (ch + block.thread_rank()) % DECOUPOLY_G_SIZE;
				atomicAdd(grad_decoupolys_g + collected_ids[j] * DECOUPOLY_G_SIZE + _ch, local_grad_decoupolys_g[_ch]);
			}
			for (int ch = 0; ch < DECOUPOLY_RANK; ch++) {
				// shuffle the atomic adds to avoid conflicts
				int _ch = (ch + block.thread_rank()) % DECOUPOLY_RANK;
				atomicAdd(grad_densities + collected_ids[j] * DECOUPOLY_RANK + _ch, local_grad_densities[_ch]);
			}
			for (int ch = 0; ch < DECOUPOLY_RANK * CHANNELS; ch++) {
				// shuffle the atomic adds to avoid conflicts
				int _ch = (ch + block.thread_rank()) % (DECOUPOLY_RANK * CHANNELS);
				atomicAdd(grad_colors + collected_ids[j] * DECOUPOLY_RANK * CHANNELS + _ch, local_grad_colors[_ch]);
			}
			#elif defined(GRAD_LOCAL_REDUCED_TO_GLOBAL)
			for (int ch = 0; ch < DECOUPOLY_V_SIZE; ch++) {
				// reduce
				float aggregated_grad = BlockReduce(temp_storage).Sum(local_grad_decoupolys_V[ch]);
				block.sync();
				if (block.thread_rank() == 0)
					shared_grad_decoupolys_V[ch] = aggregated_grad;
			}
			block.sync();
			for (int ch = block.thread_rank(); ch < DECOUPOLY_V_SIZE; ch += BLOCK_SIZE)
				atomicAdd(grad_decoupolys_V + collected_ids[j] * DECOUPOLY_V_SIZE + ch, shared_grad_decoupolys_V[ch]);
			for (int ch = 0; ch < DECOUPOLY_G_SIZE; ch++) {
				// reduce
				float aggregated_grad = BlockReduce(temp_storage).Sum(local_grad_decoupolys_g[ch]);
				block.sync();
				if (block.thread_rank() == 0)
					shared_grad_decoupolys_g[ch] = aggregated_grad;
			}
			block.sync();
			for (int ch = block.thread_rank(); ch < DECOUPOLY_G_SIZE; ch += BLOCK_SIZE)
				atomicAdd(grad_decoupolys_g + collected_ids[j] * DECOUPOLY_G_SIZE + ch, shared_grad_decoupolys_g[ch]);
			for (int ch = 0; ch < DECOUPOLY_RANK; ch++) {
				// reduce
				float aggregated_grad = BlockReduce(temp_storage).Sum(local_grad_densities[ch]);
				block.sync();
				if (block.thread_rank() == 0)
					shared_grad_densities[ch] = aggregated_grad;
			}
			block.sync();
			for (int ch = block.thread_rank(); ch < DECOUPOLY_RANK; ch += BLOCK_SIZE)
				atomicAdd(grad_densities + collected_ids[j] * DECOUPOLY_RANK + ch, shared_grad_densities[ch]);
			for (int ch = 0; ch < DECOUPOLY_RANK * CHANNELS; ch++) {
				// reduce
				float aggregated_grad = BlockReduce(temp_storage).Sum(local_grad_colors[ch]);
				block.sync();
				if (block.thread_rank() == 0)
					shared_grad_colors[ch] = aggregated_grad;
			}
			block.sync();
			for (int ch = block.thread_rank(); ch < DECOUPOLY_RANK * CHANNELS; ch += BLOCK_SIZE)
				atomicAdd(grad_colors + collected_ids[j] * DECOUPOLY_RANK * CHANNELS + ch, shared_grad_colors[ch]);
			#endif

			if (aux_grad_colors2 != nullptr) {
				for (int ch = 0; ch < CHANNELS; ch++)
					atomicMax((int*)aux_grad_colors2 + collected_ids[j] * CHANNELS + ch, __float_as_int(w_sum * dL_dout_color[ch] * dL_dout_color[ch]));
			}
			if (aux_contributions != nullptr)
				atomicMax((int*)aux_contributions + collected_ids[j], __float_as_int(w_sum));

			// If we have accumulated enough, we can stop
			if (T < 0.001f)
				done = true;
		}

		// Copy gradients to global memory
		#if defined(GRAD_SHARED_TO_GLOBAL)
		for (int j = 0; j < min(PREFETCH_BUFFER_SIZE, toDo); j++)
			for (int ch = block.thread_rank(); ch < DECOUPOLY_V_SIZE; ch += BLOCK_SIZE)
				atomicAdd(grad_decoupolys_V + collected_ids[j] * DECOUPOLY_V_SIZE + ch, grad_collected_decoupolys_V[j * DECOUPOLY_V_SIZE + ch]);
		for (int j = 0; j < min(PREFETCH_BUFFER_SIZE, toDo); j++)
			for (int ch = block.thread_rank(); ch < DECOUPOLY_G_SIZE; ch += BLOCK_SIZE)
				atomicAdd(grad_decoupolys_g + collected_ids[j] * DECOUPOLY_G_SIZE + ch, grad_collected_decoupolys_g[j * DECOUPOLY_G_SIZE + ch]);
		for (int j = 0; j < min(PREFETCH_BUFFER_SIZE, toDo); j++)
			for (int ch = block.thread_rank(); ch < DECOUPOLY_RANK; ch += BLOCK_SIZE)
				atomicAdd(grad_densities + collected_ids[j] * DECOUPOLY_RANK + ch, grad_collected_densities[j * DECOUPOLY_RANK + ch]);
		for (int j = 0; j < min(PREFETCH_BUFFER_SIZE, toDo); j++)
			for (int ch = block.thread_rank(); ch < DECOUPOLY_RANK * CHANNELS; ch += BLOCK_SIZE)
				atomicAdd(grad_colors + collected_ids[j] * DECOUPOLY_RANK * CHANNELS + ch, grad_collected_colors[j * DECOUPOLY_RANK * CHANNELS + ch]);
		#endif
	}
}


void OctreeDecoupolyRasterizer::CUDA::backward(
    const int num_nodes,
    const int active_sh_degree,
    const int num_sh_coefs,
    const int num_rendered,
    const float* background,
    const int width,
    const int height,
    const float* aabb,
    const float* positions,
    const float* decoupolys_V,
	const float* decoupolys_g,
	const float* densities,
	const float density_shift,
    const float* shs,
    const float* colors,
    const int used_rank,
    const uint8_t* depths,
    const float scale_modifier,
    const float* viewmatrix,
    const float* projmatrix,
    const float* cam_pos,
    const float tan_fovx,
    const float tan_fovy,
    const float* random_image,
    char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
    const float* out_color,
    const float* out_depth,
    const float* out_alpha,
    const float* grad_out_color,
    const float* grad_out_depth,
    const float* grad_out_alpha,
    float* grad_decoupolys_V,
	float* grad_decoupolys_g,
	float* grad_densities,
    float* grad_shs,
    float* grad_colors,
    float* aux_grad_colors2,
    float* aux_contributions
) {
	// Parrallel config (2D grid of 2D blocks)
    dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y);
    dim3 block(BLOCK_X, BLOCK_Y);

    // Recover buffers
    GeometryState geomState = GeometryState::fromChunk(geom_buffer, num_nodes);
	BinningState binningState = BinningState::fromChunk(binning_buffer, num_rendered);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	const float* color_ptr = (shs) ? geomState.colors : colors;
    CHECK_CUDA(renderBackward<<<grid, block>>>(
        imgState.ranges, binningState.point_list,
		width, height, background,
		(float3*)cam_pos, tan_fovx, tan_fovy, viewmatrix, aabb,
		positions, decoupolys_V, decoupolys_g, densities, density_shift, color_ptr, used_rank, depths, scale_modifier, random_image,
		imgState.n_contrib, imgState.t_contrib, out_color, out_depth, out_alpha,
		grad_out_color, grad_out_depth, grad_out_alpha,
        grad_decoupolys_V, grad_decoupolys_g, grad_densities, grad_colors, aux_grad_colors2, aux_contributions
    ));

	CHECK_CUDA(preprocessBackward<<<(num_nodes+255)/256, 256>>>(
		num_nodes, active_sh_degree, num_sh_coefs,
        positions, shs,
        (glm::vec3*)cam_pos, aabb,
        grad_colors, grad_shs
	));
}